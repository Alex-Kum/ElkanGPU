#include "hip/hip_runtime.h"
/* Authors: Greg Hamerly and Jonathan Drake
 * Feedback: hamerly@cs.baylor.edu
 * See: http://cs.baylor.edu/~hamerly/software/kmeans.php
 * Copyright 2014
 */

 // -lineinfo  cuda c++ comand line

#include "elkan_kmean.h"
//#include "gpufunctions.h"
#include "general_functions.h"
#include <cmath>
#include <chrono>
 //using namespace std::chrono;

#define Time 0
#define Countdistance 0

#define GPUALL 0
#if GPUALL
#define GPUA 1
#define GPUB 1
#define GPUC 1
#else
#define GPUA 0
#define GPUB 0
#define GPUC 0
#endif

#define GPUD 0

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void ElkanKmeans::update_center_dists(int threadId) {

#if GPUA
    const int n = centers->n * centers->n;
    const int blockSize = 1 * 32;
    const int numBlocks = (n + blockSize - 1) / blockSize;

    hipMemset(d_s, std::numeric_limits<double>::max(), k * sizeof(double));
    innerProd << <numBlocks, blockSize >> > (d_centerCenterDistDiv2, d_s, centers->d_data, centers->d, centers->n);

#else

    for (int c1 = 0; c1 < k; ++c1) {
        if (c1 % numThreads == threadId) {
            s[c1] = std::numeric_limits<double>::max();

            for (int c2 = 0; c2 < k; ++c2) {
                // we do not need to consider the case when c1 == c2 as centerCenterDistDiv2[c1*k+c1]
                // is equal to zero from initialization, also this distance should not be used for s[c1]
                if (c1 != c2) {
                    // divide by 2 here since we always use the inter-center
                    // distances divided by 2
                    centerCenterDistDiv2[c1 * k + c2] = sqrt(centerCenterDist2(c1, c2)) / 2.0;

                    if (centerCenterDistDiv2[c1 * k + c2] < s[c1]) {
                        s[c1] = centerCenterDistDiv2[c1 * k + c2];
                    }
                }
            }
        }
    }
#endif

}

int ElkanKmeans::runThread(int threadId, int maxIterations) {
    //std::cout << "run thread start" << std::endl;
    /*const int streamSize = 99840;
    const int nStreams = 5;*/
 /*   const int streamSize = 249600;
    const int nStreams = 2;
    hipStream_t stream[nStreams];

    for (int i = 0; i < nStreams; i++)
        hipStreamCreate(&stream[i]);*/

    int iterations = 0;
    int startNdx = start(threadId);
    int endNdx = end(threadId);

    unsigned short* closest2 = new unsigned short[endNdx];
    unsigned short* d_closest2;
    auto f = hipMalloc(&d_closest2, endNdx * sizeof(unsigned short));
    if (f != hipSuccess) {
        std::cout << "hipMalloc failed (closest2)" << std::endl;
    }

    lower = new double[n * k];
    auto g = hipMalloc(&d_lower, (n * k) * sizeof(double));
    if (g != hipSuccess) {
        std::cout << "hipMalloc failed (lower)" << std::endl;
    }
    std::fill(lower, lower + n * k, 0.0);

    bool* d_check;
    g = hipMalloc(&d_check, (k*n) * sizeof(bool));
    if (g != hipSuccess) {
        std::cout << "hipMalloc failed (check)" << std::endl;
    }

    //double* lastExactCentroid = new double[n * d];
    /*double* d_lastExactCentroid;
    g = hipMalloc(&d_lastExactCentroid, (n * d) * sizeof(double));
    if (g != hipSuccess) {
        std::cout << "hipMalloc failed (last exact)" << std::endl;
    }*/

    bool* convergedd = new bool;
    bool* d_converged;
    f = hipMalloc(&d_converged, 1 * sizeof(bool));
    if (f != hipSuccess) {
        std::cout << "hipMalloc failed (converged)" << std::endl;
    }

    converged = false;
    *convergedd = false;

#if GPUC
   /* for (int i = 0; i < n; i++) {
        for (int j = 0; j < d; j++) {
            lastExactCentroid[i * d + j] = centers->d_data[assignment[i] * d + j];
        }
    }*/

    gpuErrchk(hipMemcpy(x->d_data, x->data, (n * d) * sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_lower, lower, (n * k) * sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_upper, upper, n * sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_assignment, assignment, n * sizeof(unsigned short), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(centers->d_data, centers->data, (k * d) * sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(sumNewCenters[0]->d_data, sumNewCenters[0]->data, (k * d) * sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_clusterSize, clusterSize[0], k * sizeof(int), hipMemcpyHostToDevice));
    //gpuErrchk(hipMemcpy(d_lastExactCentroid, lastExactCentroid, (n * d) * sizeof(int), hipMemcpyHostToDevice));
   
    std::cout << "Uppper: " << upper[0] << std::endl;
    const int nC = endNdx;
    const int blockSizeC = 3 * 32;
    const int numBlocksC = (nC + blockSizeC - 1) / blockSizeC;

    const int nD = endNdx*10;
    const int blockSizeD = 3 * 32;
    const int numBlocksD = (nD + blockSizeD - 1) / blockSizeD;

    const int nM = centers->n;
    const int blockSizeM = 1 * 32;
    const int numBlocksM = (nM + blockSizeM - 1) / blockSizeM;
#endif

    while ((iterations < maxIterations) && !(*convergedd)) {
    //while ((iterations < maxIterations) && !converged) {
        *convergedd = true,
        ++iterations;

        update_center_dists(threadId);

#if GPUC    
        //gpuErrchk(hipMemcpy(d_closest2, d_assignment, n * sizeof(unsigned short), hipMemcpyDeviceToDevice));
        //elkanParallelCheck << <numBlocksD, blockSizeD >> > (x->d_data, centers->d_data, d_assignment,
         //   d_lower, d_upper, d_s, d_centerCenterDistDiv2, k, d, endNdx, d_closest2, d_clusterSize, sumNewCenters[threadId]->d_data, 0, d_check);
        elkanFunNoMove << <numBlocksC, blockSizeC >> > (x->d_data, centers->d_data, d_assignment, 
            d_lower, d_upper, d_s, d_centerCenterDistDiv2, k, d, endNdx, d_closest2, 0);
        //elkanFunNoMoveAfterCheck << <numBlocksC, blockSizeC >> > (x->d_data, centers->d_data, d_assignment, 
        //    d_lower, d_upper, d_s, d_centerCenterDistDiv2, k, d, endNdx, d_closest2, d_clusterSize, sumNewCenters[threadId]->d_data, 0, d_check);
        changeAss << <numBlocksC, blockSizeC >> > (x->d_data, d_assignment, d_closest2, d_clusterSize, sumNewCenters[threadId]->d_data, d, nC, 0);

#else
        for (int i = startNdx; i < endNdx; ++i) {
            unsigned short closest = assignment[i];
            bool r = true;

            if (upper[i] <= s[closest]) {
                continue;
            }

            for (int j = 0; j < k; ++j) {
                if (j == closest) { continue; }
                if (upper[i] <= lower[i * k + j]) { continue; }
                if (upper[i] <= centerCenterDistDiv2[closest * k + j]) { continue; }

                // ELKAN 3(a)
                if (r) {
                    upper[i] = sqrt(pointCenterDist2(i, closest));
                    lower[i * k + closest] = upper[i];
                    r = false;
                    if ((upper[i] <= lower[i * k + j]) || (upper[i] <= centerCenterDistDiv2[closest * k + j])) {
                        continue;
                    }
                }

                // ELKAN 3(b)
                lower[i * k + j] = sqrt(pointCenterDist2(i, j));
                if (lower[i * k + j] < upper[i]) {
                    closest = j;
                    upper[i] = lower[i * k + j];
                }
            }
            if (assignment[i] != closest) {
                changeAssignment(i, closest, threadId);
            }
        }
#endif

#if GPUC
        /*hipMemcpy(assignment, d_assignment, n * sizeof(unsigned short), hipMemcpyDeviceToHost);
        hipMemcpy(centers->data, centers->d_data, (k * d) * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(x->data, x->d_data, (n * d) * sizeof(double), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        verifyAssignment(iterations, startNdx, endNdx);
        hipMemcpy(d_assignment, assignment, n * sizeof(unsigned short), hipMemcpyHostToDevice);
        hipMemcpy(centers->d_data, centers->data, (k * d) * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(x->d_data, x->data, (n * d) * sizeof(double), hipMemcpyHostToDevice);
        hipDeviceSynchronize();*/

        hipMemcpy(d_converged, convergedd, 1 * sizeof(bool), hipMemcpyHostToDevice);
        elkanMoveCenter << <numBlocksM, blockSizeM >> > (d_centerMovement, d_clusterSize, centers->d_data, sumNewCenters[threadId]->d_data, d_converged, k, d, nM);
        hipMemcpy(convergedd, d_converged, 1 * sizeof(bool), hipMemcpyDeviceToHost);          

#else
        //verifyAssignment(iterations, startNdx, endNdx);
        int furthestMovingCenter = move_centers();
        converged = (0.0 == centerMovement[furthestMovingCenter]);
#endif
        // ELKAN 4, 5, AND 6
        // 
        //total_elkan_time += (std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now() - start_time));
 
        //if (!converged){
        //std::cout << "iteration: " << iterations << std::endl;
        if (!(*convergedd)) {
        //if (!converged) {
            update_bounds(startNdx, endNdx);
        }
    }

    /*hipMemcpy(assignment, d_assignment, n * sizeof(unsigned short), hipMemcpyDeviceToHost);
    for (int i = 0; i < 20; i++)
        std::cout << "assignment: " << assignment[i] << std::endl;*/
    hipFree(d_closest2);
    hipFree(d_converged);
    hipFree(d_check);
    //hipFree(d_lastExactCentroid);
    delete convergedd;
   /* for (int i = 0; i < nStreams; i++)
        hipStreamDestroy(stream[i]);*/

    return iterations;
}

void ElkanKmeans::update_bounds(int startNdx, int endNdx) {
#if GPUB
    const int n = endNdx;
    const int blockSize = 1 * 32;
    const int numBlocks = (n + blockSize - 1) / blockSize;

    updateBound << <numBlocks, blockSize >> > (x->d_data, d_lower, d_upper, d_centerMovement, d_assignment, numLowerBounds, d, k, endNdx);
#else
    for (int i = startNdx; i < endNdx; ++i) {
        upper[i] += centerMovement[assignment[i]];
        for (int j = 0; j < k; ++j) {
            lower[i * numLowerBounds + j] -= centerMovement[j];
        }
    }
#endif
}

void ElkanKmeans::initialize(Dataset const* aX, unsigned short aK, unsigned short* initialAssignment, int aNumThreads) {
    std::cout << "ElkanKmeans init" << std::endl;
    numLowerBounds = aK;
    TriangleInequalityBaseKmeans::initialize(aX, aK, initialAssignment, aNumThreads);
    std::cout << "ElkanKmeans init end" << std::endl;
    centerCenterDistDiv2 = new double[k * k];
    auto h = hipMalloc(&d_centerCenterDistDiv2, (k * k) * sizeof(double));
    if (h != hipSuccess) {
        std::cout << "hipMalloc failed (centercenterdistdiv2)" << std::endl;
    }
    std::fill(centerCenterDistDiv2, centerCenterDistDiv2 + k * k, 0.0);
}

void ElkanKmeans::free() {
    TriangleInequalityBaseKmeans::free();
    //delete[] centerCenterDistDiv2;
    hipFree(d_centerCenterDistDiv2);
    hipFree(d_lower);


    delete centerCenterDistDiv2;
    delete lower;


    centerCenterDistDiv2 = NULL;
    //delete centers;
    //centers = NULL;
}