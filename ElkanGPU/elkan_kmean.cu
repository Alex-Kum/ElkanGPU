#include "hip/hip_runtime.h"
/* Authors: Greg Hamerly and Jonathan Drake
 * Feedback: hamerly@cs.baylor.edu
 * See: http://cs.baylor.edu/~hamerly/software/kmeans.php
 * Copyright 2014
 */

 // -lineinfo  cuda c++ comand line

#include "elkan_kmean.h"
//#include "gpufunctions.h"
#include "general_functions.h"
#include <cmath>
#include <chrono>
 //using namespace std::chrono;

#define Time 0
#define Countdistance 0

#define GPUALL 0
#if GPUALL
#define GPUA 1
#define GPUB 1
#define GPUC 1
#else
#define GPUA 0
#define GPUB 0
#define GPUC 0
#endif

#define GPUD 0

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void ElkanKmeans::update_center_dists(int threadId) {

#if GPUA
    // find the inter-center distances
    for (int c1 = 0; c1 < k; ++c1) {
        s[c1] = std::numeric_limits<double>::max();
    }

    int n = centers->n * centers->n;
    int blockSize = 2 * 32;
    int numBlocks = (n + blockSize - 1) / blockSize;

    int* test = new int[n];
    int* d_test;
    hipMalloc(&d_test, n * sizeof(int));
    for (int i = 0; i < n; i++) {
        test[i] = i % 100;
    }
    hipMemcpy(d_test, test, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(centers->d_data, centers->data, (k * d) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_s, s, k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_centerCenterDistDiv2, centerCenterDistDiv2, (k * k) * sizeof(double), hipMemcpyHostToDevice);
    innerProd << <numBlocks, blockSize >> > (centerCenterDistDiv2, s, centers->data, centers->d, centers->n, d_test);
    hipDeviceSynchronize();
    hipMemcpy(centers->data, x->d_data, (k * d) * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(s, d_s, k * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(centerCenterDistDiv2, d_centerCenterDistDiv2, (k * k) * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(test, d_test, n * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < n; i++) {
        std::cout << "i: " << i << " -> " << test[i] << std::endl;
    }
    //std::cout << "COUNT A: " << count << std::endl;
    //dist2 << <1, 10 >> > (data, 0, 1, 5, 0, res);


#else

    for (int c1 = 0; c1 < k; ++c1) {
        if (c1 % numThreads == threadId) {
            s[c1] = std::numeric_limits<double>::max();

            for (int c2 = 0; c2 < k; ++c2) {
                // we do not need to consider the case when c1 == c2 as centerCenterDistDiv2[c1*k+c1]
                // is equal to zero from initialization, also this distance should not be used for s[c1]
                if (c1 != c2) {
                    // divide by 2 here since we always use the inter-center
                    // distances divided by 2
                    centerCenterDistDiv2[c1 * k + c2] = sqrt(centerCenterDist2(c1, c2)) / 2.0;

                    if (centerCenterDistDiv2[c1 * k + c2] < s[c1]) {
                        s[c1] = centerCenterDistDiv2[c1 * k + c2];
                    }
                }
            }
        }
    }
#endif

}

int ElkanKmeans::runThread(int threadId, int maxIterations) {
    std::cout << "run thread start" << std::endl;
    const int streamSize = 99840;
    const int nStreams = 5;
    hipStream_t stream[nStreams];
    hipStream_t stream1;
    hipStream_t stream2;
    for (int i = 0; i < nStreams; i++)
        hipStreamCreate(&stream[i]);
    // gpuErrchk(hipStreamCreate(&stream1));
     //gpuErrchk(hipStreamCreate(&stream2));
    int iterations = 0;

    int startNdx = start(threadId);
    int endNdx = end(threadId);
    std::cout << "endNdx : " << endNdx << std::endl;
    // bool* converged = new bool[1];
    /// bool* d_converged;
     //auto e = hipMalloc(&d_converged, 1 * sizeof(bool));
     //if (e != hipSuccess) {
    //     std::cout << "hipMalloc failed (converged)" << std::endl;
     //}


    unsigned short* closest2 = new unsigned short[endNdx];
    unsigned short* d_closest2;
    auto f = hipMalloc(&d_closest2, endNdx * sizeof(unsigned short));
    if (f != hipSuccess) {
        std::cout << "hipMalloc failed (closest2)" << std::endl;
    }

    lower = new double[n * k];
    auto g = hipMalloc(&d_lower, (n * k) * sizeof(double));
    if (g != hipSuccess) {
        std::cout << "hipMalloc failed (lower)" << std::endl;
    }
    std::fill(lower, lower + n * k, 0.0);
    converged = false;
#if GPUC
    //gpuErrchk(hipHostRegister(x->data, (n * d) * sizeof(double), hipHostRegisterDefault));
    //gpuErrchk(hipHostRegister(lower, (n * k) * sizeof(double), hipHostRegisterDefault));
    //gpuErrchk(hipHostRegister(upper, n * sizeof(double), hipHostRegisterDefault));
    //gpuErrchk(hipHostRegister(assignment, n * sizeof(unsigned short), hipHostRegisterDefault));
    //for (int i = 0; i < nStreams; i++) {
    //           int offset = i * streamSize;
    //           //std::cout << "Offset: " << offset << std::endl;
    //           //std::cout << "Number upperbounds per stream : " << (n / nStreams) << std::endl;
    //           //std::cout << (n * d) * sizeof(double) / nStreams << std::endl;
    //           //hipMemcpyAsync(&x->d_data[offset], &x->data[offset], (n * d) * sizeof(double) / nStreams, hipMemcpyHostToDevice, stream[i]);
    //           gpuErrchk(hipMemcpyAsync(&d_lower[offset], &lower[offset], (n * k) * sizeof(double) / nStreams, hipMemcpyHostToDevice, stream[i]));
    //           gpuErrchk(hipMemcpyAsync(&d_upper[offset], &upper[offset], n * sizeof(double) / nStreams, hipMemcpyHostToDevice, stream[i]));
    //           gpuErrchk(hipMemcpyAsync(&d_assignment[offset], &assignment[offset], n * sizeof(unsigned short) / nStreams, hipMemcpyHostToDevice, stream[i]));
    //       }

    gpuErrchk(hipMemcpy(x->d_data, x->data, (n * d) * sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_lower, lower, (n * k) * sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_upper, upper, n * sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_assignment, assignment, n * sizeof(unsigned short), hipMemcpyHostToDevice));

    //gpuErrchk(hipMemcpyAsync(&x->d_data[0], &x->data[0], (250000 * d) * sizeof(double), hipMemcpyHostToDevice,stream1));
   /* res = hipMemcpyAsync(d_lower, lower, (0.5 * n * k) * sizeof(double), hipMemcpyHostToDevice, stream1);
    if (res != hipSuccess)
        std::cout << "help" << std::endl;
    res = hipMemcpyAsync(d_upper, upper, 0.5 * n * sizeof(double), hipMemcpyHostToDevice, stream1);
    if (res != hipSuccess)
        std::cout << "help" << std::endl;*/
        //hipMemcpyAsync(d_assignment, assignment, 0.5 * n * sizeof(unsigned short), hipMemcpyHostToDevice, stream1);


       // gpuErrchk(hipMemcpyAsync(x->d_data + 250000, x->data + 250000, (250000 * d) * sizeof(double), hipMemcpyHostToDevice, stream1));
       // gpuErrchk(hipDeviceSynchronize());
        //gpuErrchk(hipHostUnregister(x->data));
        /*if (res != hipSuccess)
            std::cout << "help" << std::endl;
        res = hipMemcpyAsync(d_lower + 250000, lower + 250000, (0.5 * n * k) * sizeof(double), hipMemcpyHostToDevice, stream2);
        if (res != hipSuccess)
            std::cout << "help" << std::endl;
        res = hipMemcpyAsync(d_upper + 250000, upper + 250000, 0.5 * n * sizeof(double), hipMemcpyHostToDevice, stream2);
        if (res != hipSuccess)
            std::cout << "help" << std::endl;*/
            //hipMemcpyAsync(d_assignment + 250000, assignment + 250000, 0.5 * n * sizeof(unsigned short), hipMemcpyHostToDevice, stream2);


           // std::cout << "start trans" << std::endl;
            //for (int i = 0; i < nStreams; i++) {
            //    int offset = i * streamSize;
            //    //std::cout << "Offset: " << offset << std::endl;
            //    //std::cout << "Number upperbounds per stream : " << (n / nStreams) << std::endl;
            //    //std::cout << (n * d) * sizeof(double) / nStreams << std::endl;
            //    //hipMemcpyAsync(&x->d_data[offset], &x->data[offset], (n * d) * sizeof(double) / nStreams, hipMemcpyHostToDevice, stream[i]);
            //    gpuErrchk(hipMemcpyAsync(&d_lower[offset], &lower[offset], (n * k) * sizeof(double) / nStreams, hipMemcpyHostToDevice, stream[i]));
            //    gpuErrchk(hipMemcpyAsync(&d_upper[offset], &upper[offset], n * sizeof(double) / nStreams, hipMemcpyHostToDevice, stream[i]));
            //    gpuErrchk(hipMemcpyAsync(&d_assignment[offset], &assignment[offset], n * sizeof(unsigned short) / nStreams, hipMemcpyHostToDevice, stream[i]));
            //}
            //std::cout << "end trans" << std::endl;
            //hipDeviceSynchronize();
            //hipHostUnregister(x->data);
            /*

            /* gpuErrchk(hipHostRegister(centers->data, (k * d) * sizeof(double), hipHostRegisterDefault));
             gpuErrchk(hipHostRegister(s, k * sizeof(double), hipHostRegisterDefault));
             gpuErrchk(hipHostRegister(centerCenterDistDiv2, (k * k) * sizeof(double), hipHostRegisterDefault));
             gpuErrchk(hipHostRegister(closest2, n * sizeof(unsigned short), hipHostRegisterDefault));*/
#endif

    while ((iterations < maxIterations) && !(converged)) {
        //std::cout << "start iter" << iterations << std::endl;
        ++iterations;

        update_center_dists(threadId);
        synchronizeAllThreads();

#if GPUC
        //hipDeviceSynchronize();
        //hipHostUnregister(x->data);
        //hipHostUnregister(lower);
        //hipHostUnregister(upper);
        //hipHostUnregister(assignment);
        int n = endNdx;
        int blockSize = 2 * 32;
        int numBlocks = (n + blockSize - 1) / blockSize;

        //hipMemcpy(centers->d_data, centers->data, (k * d) * sizeof(double), hipMemcpyHostToDevice);
        //hipMemcpy(d_s, s, k * sizeof(double), hipMemcpyHostToDevice);
        //hipMemcpy(d_centerCenterDistDiv2, centerCenterDistDiv2, (k * k) * sizeof(double), hipMemcpyHostToDevice);
        //hipMemcpy(d_closest2, closest2, n * sizeof(unsigned short), hipMemcpyHostToDevice);

        ////hipMemcpy(d_closest2, d_assignment, n * sizeof(unsigned short), hipMemcpyDeviceToDevice);
        ////elkanFun<<<numBlocks, blockSize>>> (x->d_data, centers->d_data, d_assignment, d_lower, d_upper, d_s, d_centerCenterDistDiv2, d_clusterSize, sumNewCenters[threadId]->d_data, d_centerMovement, k, d, endNdx, numLowerBounds, d_converged, d_closest2);
        //elkanFunNoMove << <numBlocks, blockSize >> > (x->d_data, centers->d_data, d_assignment, d_lower, d_upper, d_s, d_centerCenterDistDiv2, k, d, endNdx, numLowerBounds, d_closest2, 0);
        //hipDeviceSynchronize();
        //hipMemcpy(centers->data, centers->d_data, (k * d) * sizeof(double), hipMemcpyDeviceToHost);
        //hipMemcpy(assignment, d_assignment, n * sizeof(unsigned short), hipMemcpyDeviceToHost);
        //hipMemcpy(s, d_s, k * sizeof(double), hipMemcpyDeviceToHost);
        //hipMemcpy(centerCenterDistDiv2, d_centerCenterDistDiv2, (k * k) * sizeof(double), hipMemcpyDeviceToHost);
        //hipMemcpy(closest2, d_closest2, n * sizeof(unsigned short), hipMemcpyDeviceToHost);

        hipMemcpy(centers->d_data, centers->data, (k * d) * sizeof(double), hipMemcpyHostToDevice);
       /* hipMemcpy(d_s, s, k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_centerCenterDistDiv2, centerCenterDistDiv2, (k * k) * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_closest2, closest2, n * sizeof(unsigned short), hipMemcpyHostToDevice);*/

        //for (int i = 0; i < nStreams; i++) {
        //    int offset = i * streamSize;
        //    //hipMemcpyAsync(&centers->d_data[offset], &centers->data[offset], (k * d) * sizeof(double) / nStreams, hipMemcpyHostToDevice, stream[i]);
        //    hipMemcpyAsync(&d_s[offset], &d_s[offset], k * sizeof(double) / nStreams, hipMemcpyHostToDevice, stream[i]);
        //    hipMemcpyAsync(&d_centerCenterDistDiv2[offset], &centerCenterDistDiv2[offset], (k * k) * sizeof(double) / nStreams, hipMemcpyHostToDevice, stream[i]);
        //    hipMemcpyAsync(&d_closest2[offset], &closest2[offset], n * sizeof(unsigned short) / nStreams, hipMemcpyHostToDevice, stream[i]);

        //    elkanFunNoMove << <streamSize / blockSize, blockSize, 0, stream[i] >> > (x->d_data, centers->d_data, d_assignment, d_lower, d_upper, d_s, d_centerCenterDistDiv2, k, d, endNdx, numLowerBounds, d_closest2, offset);

        //    hipMemcpyAsync(&s[offset], &d_s[offset], k * sizeof(double) / nStreams, hipMemcpyDeviceToHost, stream[i]);
        //    hipMemcpyAsync(&centerCenterDistDiv2[offset], &d_centerCenterDistDiv2[offset], (k * k) * sizeof(double) / nStreams, hipMemcpyDeviceToHost, stream[i]);
        //    hipMemcpyAsync(&closest2[offset], &d_closest2[offset], n * sizeof(unsigned short) / nStreams, hipMemcpyDeviceToHost, stream[i]);
        //    //hipMemcpyAsync(&centerCenterDistDiv2[offset], &d_centerCenterDistDiv2[offset], (k * k) * sizeof(double) / nStreams, hipMemcpyDeviceToHost, stream[i]);
        //    hipMemcpyAsync(&assignment[offset], &d_assignment[offset], n * sizeof(unsigned short) / nStreams, hipMemcpyDeviceToHost, stream[i]);
        //    //hipMemcpyAsync(&centers->data[offset], &centers->d_data[offset], (k * d) * sizeof(double) / nStreams, hipMemcpyDeviceToHost, stream[i]);
        //}

        for (int i = 0; i < nStreams; i++) {
            int offset = i * streamSize;
            //hipMemcpyAsync(&centers->d_data[offset], &centers->data[offset], (k * d) * sizeof(double) / nStreams, hipMemcpyHostToDevice, stream[i]);
            hipMemcpyAsync(&d_s[offset], &d_s[offset], k * sizeof(double) / nStreams, hipMemcpyHostToDevice, stream[i]);
            hipMemcpyAsync(&d_centerCenterDistDiv2[offset], &centerCenterDistDiv2[offset], (k * k) * sizeof(double) / nStreams, hipMemcpyHostToDevice, stream[i]);
            hipMemcpyAsync(&d_closest2[offset], &closest2[offset], n * sizeof(unsigned short) / nStreams, hipMemcpyHostToDevice, stream[i]);
        }

        for (int i = 0; i < nStreams; i++) {
            int offset = i * streamSize;
            elkanFunNoMove << <streamSize / blockSize, blockSize, 0, stream[i] >> > (x->d_data, centers->d_data, d_assignment, d_lower, d_upper, d_s, d_centerCenterDistDiv2, k, d, endNdx, numLowerBounds, d_closest2, offset);
        }

        for (int i = 0; i < nStreams; i++) {
            int offset = i * streamSize;
            hipMemcpyAsync(&s[offset], &d_s[offset], k * sizeof(double) / nStreams, hipMemcpyDeviceToHost, stream[i]);
            hipMemcpyAsync(&centerCenterDistDiv2[offset], &d_centerCenterDistDiv2[offset], (k * k) * sizeof(double) / nStreams, hipMemcpyDeviceToHost, stream[i]);
            hipMemcpyAsync(&closest2[offset], &d_closest2[offset], n * sizeof(unsigned short) / nStreams, hipMemcpyDeviceToHost, stream[i]);
            //hipMemcpyAsync(&centerCenterDistDiv2[offset], &d_centerCenterDistDiv2[offset], (k * k) * sizeof(double) / nStreams, hipMemcpyDeviceToHost, stream[i]);
            hipMemcpyAsync(&assignment[offset], &d_assignment[offset], n * sizeof(unsigned short) / nStreams, hipMemcpyDeviceToHost, stream[i]);
            //hipMemcpyAsync(&centers->data[offset], &centers->d_data[offset], (k * d) * sizeof(double) / nStreams, hipMemcpyDeviceToHost, stream[i]);
        }



        //elkanFunNoMove << <numBlocks, blockSize >> > (x->d_data, centers->d_data, d_assignment, d_lower, d_upper, d_s, d_centerCenterDistDiv2, k, d, endNdx, numLowerBounds, d_closest2, 0);
        hipDeviceSynchronize();

        hipMemcpy(centers->data, centers->d_data, (k * d) * sizeof(double), hipMemcpyDeviceToHost);
        /*hipMemcpy(assignment, d_assignment, n * sizeof(unsigned short), hipMemcpyDeviceToHost);

        hipMemcpy(s, d_s, k * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(centerCenterDistDiv2, d_centerCenterDistDiv2, (k * k) * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(closest2, d_closest2, n * sizeof(unsigned short), hipMemcpyDeviceToHost);*/

        /* int count = 0;
         for (int i = 0; i < n; i++) {
             count += test[i];
             std::cout << "i: " << i << " -> " << test[i] << std::endl;
         }*/
         //std::cout << "COUNT: " << count << std::endl;

         //elkanFunNoMove<<<numBlocks, blockSize>>> (x->data, centers->data, assignment, lower, upper, s, centerCenterDistDiv2, k, d, endNdx, numLowerBounds, converged, closest2);
         //hipDeviceSynchronize();

         //changeAss<<<numBlocks, blockSize >>>(x->data, assignment, closest2, clusterSize, sumNewCenters[threadId]->data, d, endNdx);
         //hipDeviceSynchronize();

        for (int i = startNdx; i < endNdx; ++i) {
            if (assignment[i] != closest2[i]) {
                changeAssignment(i, closest2[i], threadId);
            }
        }
#else
        for (int i = startNdx; i < endNdx; ++i) {
            //std::cout << d << "\n";
            unsigned short closest = assignment[i];
            bool r = true;

            if (upper[i] <= s[closest]) {
                continue;
            }

            for (int j = 0; j < k; ++j) {
                if (j == closest) { continue; }
                if (upper[i] <= lower[i * k + j]) { continue; }
                if (upper[i] <= centerCenterDistDiv2[closest * k + j]) { continue; }

                // ELKAN 3(a)
                if (r) {
                    upper[i] = sqrt(pointCenterDist2(i, closest));
                    lower[i * k + closest] = upper[i];
                    r = false;
                    if ((upper[i] <= lower[i * k + j]) || (upper[i] <= centerCenterDistDiv2[closest * k + j])) {
                        continue;
                    }
                }

                // ELKAN 3(b)
                lower[i * k + j] = sqrt(pointCenterDist2(i, j));
                if (lower[i * k + j] < upper[i]) {
                    closest = j;
                    upper[i] = lower[i * k + j];
                }
            }
            if (assignment[i] != closest) {
                changeAssignment(i, closest, threadId);
            }
        }
#endif


        //verifyAssignment(iterations, startNdx, endNdx);

        // ELKAN 4, 5, AND 6
        synchronizeAllThreads();

        if (threadId == 0) {
            int furthestMovingCenter = move_centers();
            converged = (0.0 == centerMovement[furthestMovingCenter]);
            //std::cout << "Furthest Movement: " << centerMovement[furthestMovingCenter] << " (center " << furthestMovingCenter << ")" <<  std::endl;
        }

        synchronizeAllThreads();
        //total_elkan_time += (std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now() - start_time));
        if (!converged) {
            update_bounds(startNdx, endNdx);
        }
        else {
            std::cout << "Iterations: " << iterations << "\n";
        }
    }

    //hipFree(d_converged);
    hipFree(d_closest2);
    for (int i = 0; i < nStreams; i++)
        hipStreamDestroy(stream[i]);
    //hipStreamDestroy(stream1);
    //hipStreamDestroy(stream2);

    std::cout << "ITERATIONEN: " << iterations << std::endl;
    return iterations;
}

void ElkanKmeans::update_bounds(int startNdx, int endNdx) {
#if GPUB
    const int n = endNdx;
    const int blockSize = 2 * 32;
    const int numBlocks = (n + blockSize - 1) / blockSize;
    hipMemcpy(d_assignment, assignment, n * sizeof(unsigned short), hipMemcpyHostToDevice);
    hipMemcpy(d_centerMovement, centerMovement, k * sizeof(double), hipMemcpyHostToDevice);

    updateBound << <numBlocks, blockSize >> > (d_lower, d_upper, d_centerMovement, d_assignment, numLowerBounds, k, endNdx);
    hipDeviceSynchronize();

    hipMemcpy(centerMovement, d_centerMovement, k * sizeof(double), hipMemcpyDeviceToHost);
#else
    for (int i = startNdx; i < endNdx; ++i) {
        upper[i] += centerMovement[assignment[i]];
        for (int j = 0; j < k; ++j) {
            lower[i * numLowerBounds + j] -= centerMovement[j];
        }
    }
#endif
}

void ElkanKmeans::initialize(Dataset const* aX, unsigned short aK, unsigned short* initialAssignment, int aNumThreads) {
    std::cout << "ElkanKmeans init" << std::endl;
    numLowerBounds = aK;
    TriangleInequalityBaseKmeans::initialize(aX, aK, initialAssignment, aNumThreads);
    std::cout << "ElkanKmeans init end" << std::endl;
    centerCenterDistDiv2 = new double[k * k];
    auto h = hipMalloc(&d_centerCenterDistDiv2, (k * k) * sizeof(double));
    if (h != hipSuccess) {
        std::cout << "hipMalloc failed (centercenterdistdiv2)" << std::endl;
    }
    std::fill(centerCenterDistDiv2, centerCenterDistDiv2 + k * k, 0.0);
}

void ElkanKmeans::free() {
    TriangleInequalityBaseKmeans::free();
    //delete[] centerCenterDistDiv2;
    hipFree(d_centerCenterDistDiv2);
    hipFree(d_lower);
    delete centerCenterDistDiv2;
    delete lower;

    centerCenterDistDiv2 = NULL;
    //delete centers;
    //centers = NULL;
}


/*CPU:

for (int i = 0; i < n; i++) {
    test[i] = i % 100;
    //closest2[i] = assignment[i];
}

hipMemcpy(d_assignment, assignment, n * sizeof(unsigned short), hipMemcpyHostToDevice);
hipMemcpy(d_closest2, closest2, n * sizeof(unsigned short), hipMemcpyHostToDevice);
hipMemcpy(d_test, test, n * sizeof(int), hipMemcpyHostToDevice);

elkanFunNoMove << <numBlocks, blockSize >> > (x->d_data, centers->d_data, d_assignment, d_lower, d_upper, d_s, d_centerCenterDistDiv2,
    k, d, endNdx, numLowerBounds, d_closest2, d_test);
hipDeviceSynchronize();

hipMemcpy(assignment, d_assignment, n * sizeof(unsigned short), hipMemcpyDeviceToHost);
hipMemcpy(closest2, d_closest2, n * sizeof(unsigned short), hipMemcpyDeviceToHost);
hipMemcpy(test, d_test, n * sizeof(int), hipMemcpyDeviceToHost);

for (int i = 0; i < n; i++) {
    std::cout << "i: " << i << " -> " << test[i] << std::endl;
}

GPU:
__global__ void elkanFunNoMove(double* data, double* center, unsigned short* assignment, double* lower, double* upper,
    double* s, double* centerCenterDistDiv2, int k, int dim, int n, int numlower, unsigned short* closest2, int* test) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        test[i] = 5;
        //closest2[i] = closest2[i] - 1;           1
        //closest2[i] = 1;                         2
        //closest2[i] = assignment[i];             3
        ...
*/
