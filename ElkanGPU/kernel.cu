#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "Dataset.h"
//#include "gpufunctions.h"
#include <hipblas.h>
#include <random>
#include <fstream>
#include <string>

#include "general_functions.h"
#include "kmeans.h"
#include "elkan_kmean.h"
#include "ham_elkan.h"
#include "ham_elkanFB.h"
#include "ham_elkanMO.h"

#include "yy_kmean.h"
#include "FB1_elkan_kmeans.h"
#include "MO_elkan_kmeans.h"
#include "combinedElkan.h"

#include <fstream>
#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    printf("add: %i\n", a[i]);
    c[i] = a[i] + b[i];
}

__global__ void multiplyKernel(float* a, float* b, float* c) {
    int i = threadIdx.x;
    //printf("ahhh");
    if (i < 5) {
        c[i] = a[i] * b[i];
    }
}

__global__ void setTestt(int* test, unsigned short* arr1, unsigned short* arr2) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < 1000) {
        arr2[i] = arr1[i];
        test[i] = 5;
    }
}

Dataset* loadDataset(std::string const& filename, int n, int d) {
    std::string number;
    Dataset* x = nullptr;
    fstream file;

    file.open(filename, ios::in);
    x = new Dataset(n, d);
    int i = 0;
    while (getline(file, number, ','))
    {
        x->data[i] = atof(number.c_str());       
        i++;
        if (i >= n * d)
            break;
    }   
    file.close();
    return x;



    //std::ifstream input(filename.c_str());

    //int n, d;
    //input >> n >> d;

    //Dataset* x = new Dataset(n, d);

    //double* dataTMP = new double[n * d];


    ////double* copyP1 = x->data;
    //for (int i = 0; i < n * d; ++i) {
    //    input >> x->data[i];
    //    //copyP1++;
    //}
    //return x;
}

int importPoints(
    PointInfo* pointInfo,
    DTYPE* pointData,
    const int numPnt,
    const int numDim)
{
    std::string number;
    fstream file;
    //file.open("file.txt", ios::in);
    //file.open("gassensor_clean.data", ios::in);
    file.open("KEGGNetwork_clean.data", ios::in);    
    //file.open("USCensus_clean.data", ios::in);
    int i = 0;
    while (getline(file, number, ','))
    {
        pointData[i] = atof(number.c_str());
        i++;
    }

    for (int j = 0; j < numPnt; j++) {
        pointInfo[j].centroidIndex = -1;
        pointInfo[j].oldCentroid = -1;
        pointInfo[j].uprBound = INFINITY;
    }
    return 0;
}

Dataset* load_randDataset(int n, int d) {
    bool createNew = false;
    std::string number;
    Dataset* x = nullptr;
    fstream file;

    if (!createNew) {
        file.open("file.txt", ios::in);
        x = new Dataset(n, d);
        int i = 0;
        while (getline(file, number, ','))
        {
            x->data[i] = atof(number.c_str());
            i++;
        }
        std::cout << "GELESEN: " << i << std::endl;
    }
    else {
        file.open("file.txt", ios::out | ios::trunc);
        file << (double)rand() / (double)RAND_MAX;

        for (int i = 1; i < n * d; ++i) {
            file << ",";
            file << (double)rand() / (double)RAND_MAX;
            // x->data[i] = dis(gen);
            //x->data[i] = (double)rand() / (double)RAND_MAX;
        }
    }
    file.close();
    return x;
}

std::random_device rd;
std::mt19937 gen(rd());
std::uniform_real_distribution<> dis(0.1, 100.0);

int main(){
    hipSetDevice(0);   
    //int num = 5;
    //float* data1 = new float[num];
    //float* d_data1;
    //hipMalloc(&d_data1, num * sizeof(float));
    //float* data2 = new float[num];
    //float* d_data2;
    //hipMalloc(&d_data2, num * sizeof(float));
    //float* data3 = new float[num];
    //float* d_data3;
    //hipMalloc(&d_data3, num * sizeof(float));

    //for (int i = 0; i < 5; i++)
    //    data1[i] = i;

    //for (int i = 0; i < 5; i++)
    //    data2[i] = 5 - i;
    //data2[2] = 2;
    //data2[3] = 3;

    //hipMemcpy(d_data1, data1, num * sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy(d_data2, data2, num * sizeof(float), hipMemcpyHostToDevice);
    //multiplyKernel << <1, 5 >> > (d_data1, d_data2, d_data3);
    //hipDeviceSynchronize();
    //hipMemcpy(data3, d_data3, num * sizeof(float), hipMemcpyDeviceToHost);
    //hipDeviceSynchronize();
    //for (int i = 0; i < 5; i++) {
    //    std::cout << "Nicht-Managed:" << data1[i] << " * " << data2[i] << " = " << data3[i] << std::endl;
    //    //std::cout << arr1[i] << " dot " << arr2[i] << std::endl;
    //}
    //hipFree(d_data1);
    //hipFree(d_data2);
    //hipFree(d_data3);
    //delete data1;
    //delete data2;
    //delete data3;
    //warumGehtNichts();


   // hipDeviceProp_t prop;
   // hipGetDeviceProperties(&prop, 0);

    
    //int clusters[] = { 4,16,64,265 };
    //Dataset* x = loadDataset("KEGGNetwork_clean.data", 65554, 28);
    Dataset* x = loadDataset("USCensus_clean.data", 2458285, 68);
    //Dataset* x = loadDataset("gassensor_clean.data", 13910, 128);
    if (x == nullptr) {
        cout << "Dataset generated" << endl;
        return 0;
    }
    cout << "Dataset loaded" << endl;

    vector< std::chrono::duration<double>> results;
        int k = 64;
        cout << "k: " << k << endl;
        Dataset* initialCenters = init_centers(*x, k);
        for (int i = 0; i < 1; i++) {                        
            // * alg = new HamElkan();
            //HamElkan* alg = new HamElkan();
            //ElkanKmeans* alg = new ElkanKmeans();
            //FB1_ElkanKmeans* alg = new FB1_ElkanKmeans();
            MO_ElkanKmeans* alg = new MO_ElkanKmeans();
            //HamElkanFB* alg = new HamElkanFB();
            //HamElkanMO* alg = new HamElkanMO();
            //CO_ElkanKmeans* alg = new CO_ElkanKmeans();
            std::cout << "Alg: " << alg->getName() << std::endl;
            //Dataset* x = load_dataset("C:\\Users\\Admin\\Desktop\\MASTER\\skin_nonskin.txt");
            //Dataset* x = loadDataset("file.txt", 499200, 100);   

            
            unsigned short* assignment = new unsigned short[x->n];

            assign(*x, *initialCenters, assignment);
            alg->initialize(x, k, assignment, 1);
            //std::cout << "assignment 0" << assignment[0] << std::endl;

            auto start = std::chrono::system_clock::now();
            std::cout << "alg run start" << std::endl;
            int iterations = alg->run(5000);
            std::cout << "alg run end" << std::endl;
            std::cout << "Iterations: " << iterations << std::endl;
            auto end = std::chrono::system_clock::now();
            std::chrono::duration<double> elapsed_seconds = end - start;
            results.push_back(elapsed_seconds);
            std::cout << "Sekunden: " << elapsed_seconds.count() << "\n";

            delete[] assignment;
            delete alg;    
        }
        delete initialCenters;
    
    hipDeviceSynchronize();

    int counter = 0;
    for (auto& e : results) {
        cout << e.count() << ", ";
        counter++;        
    }
    cout << endl;
   
    delete x;
    hipDeviceReset();

     //!____________________________________________________________________________________________________________________________________________________
    ///*const int numPnt = 499200;
    //const int numCent = 10;
    //const int numDim = 100;*/
    //const int numPnt = 13910;    
    //const int numCent = 100;
    //const int numDim = 128;
    ///*const int numPnt = 65554;
    //const int numCent = 100;
    //const int numDim = 28;*/
    ///*const int numPnt = 2458285;
    //const int numCent = 100;
    //const int numDim = 68;*/
    //const int numGrp = 10;


    
   // const  int numThread = 1;
   // const int maxIter = 5000;
   // const  int numGPU = 1;
   // double runtime;
   // int writeCentFlag = 0;
   // int writeAssignFlag = 0;
   // int writeTimeFlag = 0;
   // char* writeCentPath;
   // char* writeAssignPath;
   // char* writeTimePath;
   // int countFlag = 0;
   // unsigned long long int calcCount = 0;

   // std::cout << "bevor pointInfo" << std::endl;
   // //import and create dataset
   // PointInfo* pointInfo = (PointInfo*)malloc(sizeof(PointInfo) * numPnt);
   // //PointInfo* pointInfo = new PointInfo[numPnt];
   // DTYPE* pointData = (DTYPE*)malloc(sizeof(DTYPE) * numPnt * numDim);


   // if (importPoints(pointInfo, pointData, numPnt, numDim))
   // {
   //     // signal erroneous exit
   //     printf("\nERROR: could not import the dataset, please check file location. Exiting program.\n");
   //     free(pointInfo);
   //     free(pointData);
   //     return 1;
   // }

   // std::cout << "bevor centInfo" << std::endl;
   // CentInfo* centInfo = (CentInfo*)malloc(sizeof(CentInfo) * numCent);
   // DTYPE* centData = (DTYPE*)malloc(sizeof(DTYPE) * numCent * numDim);

   // // generate centroid data using dataset points
    //if (generateCentWithDataSame(centInfo, centData, pointData, numCent, numPnt, numDim))
   // {
   //     // signal erroneous exit
   //     printf("\nERROR: Could not generate centroids. Exiting program.\n");
   //     free(pointInfo);
   //     free(pointData);
   //     free(centInfo);
   //     free(centData);
   //     return 1;
   // }
   // unsigned int ranIter;
   // std::cout << "alg run start" << std::endl;
   // std::cout << "numPnt: " << numPnt << std::endl;
   // std::cout << "numCent: " << numCent << std::endl;
   // std::cout << "numGrp: " << numGrp << std::endl;
   // std::cout << "numDim: " << numDim << std::endl;
   // std::cout << "maxIter: " << maxIter << std::endl;
   // std::cout << "numGPU: " << numGPU << std::endl;
   // /*std::cout << "pInfo: " << pointInfo[40000].centroidIndex << std::endl;
   // std::cout << "cInfo: " << centInfo[4].count << std::endl;
   // std::cout << "pData: " << pointData[400000] << std::endl;
   // std::cout << "cData: " << centData[40] << std::endl;*/
   // auto start = std::chrono::system_clock::now();
   // warmupGPU(numGPU);
   // //runtime = warumGehtNichts();
   ///* runtime =
   //     startLloydOnGPU(pointInfo, centInfo, pointData, centData,
   //         numPnt, numCent, numDim, maxIter, numGPU, &ranIter);*/
   ///* runtime =
   //     startLloydOnCPU(pointInfo, centInfo, pointData, centData,
   //         numPnt, numCent, numDim, 1, maxIter, &ranIter);*/
   // runtime =
   //     startSimpleOnGPU(pointInfo, centInfo, pointData, centData,
   //         numPnt, numCent, numGrp, numDim, maxIter, numGPU,
   //         &ranIter);

   // /*runtime =
   //     startSuperOnGPU(pointInfo, centInfo, pointData, centData,
   //         numPnt, numCent, numDim, maxIter, numGPU, &ranIter);*/
   ///* runtime =
   //     startSimpleOnCPU(pointInfo, centInfo, pointData, centData, numPnt,
   //         numCent, numGrp, numDim, numThread, maxIter, &ranIter);*/

   // auto end = std::chrono::system_clock::now();
   // std::cout << "alg run end" << std::endl;
   // std::chrono::duration<double> elapsed_seconds = end - start;
   // std::cout << "Sekunden: " << elapsed_seconds.count() << "\n";
   // std::cout << "ITERATIONS: " << ranIter << std::endl;

   // for (int i = 0; i < 20; i++) {
   //     std::cout << "Assignment: " << i << " -> " << pointInfo[i].centroidIndex << std::endl;
   // }
   // free(pointData);
   // free(centData);
   // free(pointInfo);
   // free(centInfo);
   //
   //  // hipDeviceReset must be called before exiting in order for profiling and
   //  // tracing tools such as Nsight and Visual Profiler to show complete traces.
   // auto res = hipDeviceReset();
   // if (res != hipSuccess) {
   //     fprintf(stderr, "hipDeviceReset failed!");
   //     return 1;
   // }

    return 0;
}
