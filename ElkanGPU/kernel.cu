#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "Dataset.h"
//#include "gpufunctions.h"
#include <hipblas.h>
#include <random>
#include <fstream>
#include <string>

#include "general_functions.h"
#include "kmeans.h"
#include "elkan_kmean.h"
//#include "FB1_elkan_kmeans.h"
//#include "MO_elkan_kmeans.h"

#include <fstream>
#include <stdio.h>
#include <iostream>

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    printf("add: %i\n", a[i]);
    c[i] = a[i] + b[i];
}

__global__ void multiplyKernel(const float* a, const float* b, float* c) {
    int i = threadIdx.x;
    if (i < 5) {
        c[i] = a[i] * b[i];
    }
}

__global__ void setTestt(int* test, unsigned short* arr1, unsigned short* arr2) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < 1000) {
        arr2[i] = arr1[i];
        test[i] = 5;        
    }
}

Dataset* load_dataset(std::string const& filename) {
    std::ifstream input(filename.c_str());

    int n, d;
    input >> n >> d;

    Dataset* x = new Dataset(n, d);

    double* dataTMP = new double[n * d];


    //double* copyP1 = x->data;
    for (int i = 0; i < n * d; ++i) {
        input >> x->data[i];
        //copyP1++;
    }   
    return x;
}

Dataset* load_randDataset(int n, int d) {
    bool createNew = false;
    std::string number;
    Dataset* x = nullptr;
    fstream file;
    
    if (!createNew) {
        file.open("file.txt", ios::in);
        x = new Dataset(n, d);
        int i = 0;
        while (getline(file, number, ','))
        {
            x->data[i] = atof(number.c_str());
            i++;
        }
    }
    else {
        file.open("file.txt", ios::out | ios::trunc);
        file << (double)rand() / (double)RAND_MAX;
        
        for (int i = 1; i < n * d; ++i) {
            file << ",";
            file << (double)rand() / (double)RAND_MAX;
            // x->data[i] = dis(gen);
            //x->data[i] = (double)rand() / (double)RAND_MAX;
        }
    }
    file.close();
    return x;
}

std::random_device rd;
std::mt19937 gen(rd());
std::uniform_real_distribution<> dis(0.1, 100.0);

int main()
{           
    hipSetDevice(0);

    /*const int streamSize = 200;
    const int nStreams = 5;
    hipStream_t stream[nStreams];
    for (int i = 0; i < nStreams; i++)
        hipStreamCreate(&stream[i]);

    const int n = 1000;
    int* arr1 = new int[n];
    int* d_arr1;
    auto f = hipMalloc(&d_arr1, n * sizeof(int));
    if (f != hipSuccess) {
        std::cout << "hipMalloc failed (arr1)" << std::endl;
    }

    int* arr2 = new int[n];
    int* d_arr2;
    f = hipMalloc(&d_arr2, n * sizeof(int));
    if (f != hipSuccess) {
        std::cout << "hipMalloc failed (arr2)" << std::endl;
    }

    int* arr3 = new int[n];
    int* d_arr3;
    f = hipMalloc(&d_arr3, n * sizeof(int));
    if (f != hipSuccess) {
        std::cout << "hipMalloc failed (arr2)" << std::endl;
    }

    for (int i = 0; i < n; i++) {
        arr1[i] = i;
        arr2[i] = i;
    }

    int blockSize = 1 * 32;
    int numBlocks = (n + blockSize - 1) / blockSize;
    hipHostRegister(arr1, n * sizeof(int), hipHostRegisterDefault);
    hipHostRegister(arr2, n * sizeof(int), hipHostRegisterDefault);
    hipHostRegister(arr3, n * sizeof(int), hipHostRegisterDefault);

    for (int i = 0; i < nStreams; i++) {
        int offset = i * streamSize;
        hipMemcpyAsync(&d_arr1[offset], &arr1[offset], n * sizeof(double) / nStreams, hipMemcpyHostToDevice, stream[i]);
        hipMemcpyAsync(&d_arr2[offset], &arr2[offset], n  * sizeof(double) / nStreams, hipMemcpyHostToDevice, stream[i]);
        hipMemcpyAsync(&d_arr3[offset], &arr3[offset], n * sizeof(double) / nStreams, hipMemcpyHostToDevice, stream[i]);
    }
    hipMemcpy(d_arr1, arr1, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_arr2, arr2, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_arr3, arr3, n * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    hipHostUnregister(arr1);
    hipHostUnregister(arr2);
    hipHostUnregister(arr3);
    addKernel << <1, n >> > (d_arr3, d_arr1, d_arr2);
    hipDeviceSynchronize();

    hipMemcpy(arr1, d_arr1, n * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(arr2, d_arr2, n * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(arr3, d_arr3, n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        cout << arr1[i] << " + " << arr2[i] << " = " << arr3[i] << endl;
    }*/

   /* const int n = 1000;
    unsigned short* arr1 = new unsigned short[n];
    unsigned short* d_arr1;
    auto f = hipMalloc(&d_arr1, n * sizeof(unsigned short));
    if (f != hipSuccess) {
        std::cout << "hipMalloc failed (arr1)" << std::endl;
    }

    unsigned short* arr2 = new unsigned short[n];
    unsigned short* d_arr2;
    f = hipMalloc(&d_arr2, n * sizeof(unsigned short));
    if (f != hipSuccess) {
        std::cout << "hipMalloc failed (arr2)" << std::endl;
    }

    int* test = new int[n];
    int* d_test;
    f = hipMalloc(&d_test, n * sizeof(int));
    if (f != hipSuccess) {
        std::cout << "hipMalloc failed (arr2)" << std::endl;
    }

    for (int i = 0; i < n; i++) {
        arr1[i] = 0;
        arr2[i] = 0;
        test[i] = i % 100;
    }

    int blockSize = 2 * 32;
    int numBlocks = (n + blockSize - 1) / blockSize;

    hipMemcpy(d_test, test, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_arr1, arr1, n * sizeof(unsigned short), hipMemcpyHostToDevice);
    hipMemcpy(d_arr2, arr2, n * sizeof(unsigned short), hipMemcpyHostToDevice);

    setTestt << <numBlocks, blockSize >> > (d_test, d_arr1, d_arr2);
    hipDeviceSynchronize();
    hipMemcpy(test, d_test, n * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(arr1, d_arr1, n * sizeof(unsigned short), hipMemcpyDeviceToHost);
    hipMemcpy(arr2, d_arr2, n * sizeof(unsigned short), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        std::cout << "i: " << i << " -> " << test[i] << std::endl;
    }*/

    
   hipblasHandle_t cublas_handle;
    hipblasStatus_t stat;
    stat = hipblasCreate(&cublas_handle);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    //cout << "cudaenginecount: " << prop.asyncEngineCount << endl;
    int k = 10;
    ElkanKmeans* alg = new ElkanKmeans();
    //FB1_ElkanKmeans* alg = new FB1_ElkanKmeans();
    //MO_ElkanKmeans* alg = new MO_ElkanKmeans();
    std::cout << "Alg: " << alg->getName() << std::endl;
    //Dataset* x = load_dataset("C:\\Users\\Admin\\Desktop\\MASTER\\skin_nonskin.txt");
    Dataset* x = load_randDataset(499200,10);
    if (x == nullptr) {
        cout << "Dataset generated" << endl;
        return 0;
    }
    cout << "Dataset loaded" << endl;
    //auto alg = make_unique<ElkanKmeans>(ElkanKmeans());
    Dataset* initialCenters = init_centers(*x, k);
    unsigned short* assignment = new unsigned short[x->n];
    unsigned short* d_assignment;

    //std::cout << "d_assignment malloc n: " << x->n << std::endl;
   
    assign(*x, *initialCenters, assignment);
    alg->initialize(x, k, assignment, 1);

    auto start = std::chrono::system_clock::now();
    std::cout << "alg run start" << std::endl;
    alg->run(5000);
    std::cout << "alg run end" << std::endl;
    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_seconds = end - start;
    std::cout << "Sekunden: " << elapsed_seconds.count() << "\n";
    std::cout << std::numeric_limits<double>::max() << std::endl;
    //auto x = Dataset(5, 4);
    //x.fill(3.0);
    //x.print();
    hipDeviceSynchronize();
    //hipFree(d_assignment);
    delete assignment;
    delete alg;
    delete x;
    //hipDeviceReset();

    /*int num = 5;
    float* data1 = new float[num];
    float* d_data1;
    hipMalloc(&d_data1, num * sizeof(float));
    float* data2 = new float[num];
    float* d_data2;
    hipMalloc(&d_data2, num * sizeof(float));
    float* data3 = new float[num];
    float* d_data3;
    hipMalloc(&d_data3, num * sizeof(float));

    for (int i = 0; i < 5; i++)
        data1[i] = i;

    for (int i = 0; i < 5; i++)
        data2[i] = 5 - i;
    data2[2] = 2;
    data2[3] = 3;

    hipMemcpy(d_data1, data1, num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_data2, data2, num * sizeof(float), hipMemcpyHostToDevice);
   // hipMemcpy(d_data3, data3, num * sizeof(float), hipMemcpyHostToDevice);

    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    const int s = 5;
    float* arr1;
    float* arr2;
    float* arr3;
    
    hipMallocManaged(&arr1, 5 * sizeof(float));
    hipMallocManaged(&arr2, 5 * sizeof(float));
    hipMallocManaged(&arr3, 5 * sizeof(float));
    double* data;
    double* res;
    hipMallocManaged(&data, 10 * sizeof(double));
    hipMallocManaged(&res, 1 * sizeof(double));

    for (int i = 0; i < 5; i++)
        arr1[i] = i;

    for (int i = 0; i < 5; i++)
        arr2[i] = 5-i;
    arr2[2] = 2;
    arr2[3] = 3;

    for (int i = 0; i < 5; i++) {
        data[i] = arr1[i];
    }
    for (int i = 0; i < 5; i++) {
        data[i+5] = arr2[i];
    }
    for (int i = 0; i < 10; i++)
        cout << "i: " << i << " " << data[i] << endl;

    multiplyKernel<<<1,5>>>(arr1, arr2, arr3);
    //mult <<<1, s>>> (arr1, arr2, arr3);
    //dist2<<<1, 10>>> (data, 0, 1, 5, 0, res);
    //hipblasSdot(cublas_handle, 5, arr1, 1, arr2, 1, &arr3[0]);
    hipDeviceSynchronize();
    multiplyKernel << <1, 5 >> > (d_data1, d_data2, d_data3);
    hipDeviceSynchronize();
    //hipMemcpy(data1, d_data1, num * sizeof(float), hipMemcpyDeviceToHost);
    //hipMemcpy(data2, d_data2, num * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(data3, d_data3, num * sizeof(float), hipMemcpyDeviceToHost);
    //hipDeviceSynchronize();
    //std::cout << "ERGEBNIS: " << *res << std::endl;

    
    for (int i = 0; i < 5; i++) {
        std::cout << "Managed:" << arr1[i] << " * " << arr2[i] << " = " << arr3[i] << std::endl;
        //std::cout << arr1[i] << " dot " << arr2[i] << std::endl;
    }

    for (int i = 0; i < 5; i++) {
        std::cout << "Nicht-Managed:" << data1[i] << " * " << data2[i] << " = " << data3[i] << std::endl;
        //std::cout << arr1[i] << " dot " << arr2[i] << std::endl;
    }
    //std::cout << "ERGEBNIS: " << *res << std::endl;
   // std::cout << "= " << arr3[0] << std::endl;
    hipFree(arr1);
    hipFree(arr2);
    hipFree(arr3);
    //hipblasDestroy(cublas_handle);
    std::cout << "------------" << std::endl;

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);
     */
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    auto res = hipDeviceReset();
    if (res != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
