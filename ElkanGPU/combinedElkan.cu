#include "hip/hip_runtime.h"
/* Authors: Greg Hamerly and Jonathan Drake
 * Feedback: hamerly@cs.baylor.edu
 * See: http://cs.baylor.edu/~hamerly/software/kmeans.php
 * Copyright 2014
 */

#include "CO_elkan_kmeans.h"
#include "general_functions.h"
#include "gpufunctions.h"
#include <cmath>
#include <chrono>
 //using namespace std::chrono;

#define Time 0
#define Countdistance 0
#define GPUA 1
#define GPUB 1
#define GPUC 1


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void CO_ElkanKmeans::update_center_dists(int threadId) {
#if GPUA
    const int n = centers->n * centers->n;
    const int blockSize = 3 * 32;
    const int numBlocks = (n + blockSize - 1) / blockSize;
    hipMemset(d_s, std::numeric_limits<double>::max(), k * sizeof(double));
    if (iterations > changeIter){
        innerProdMO << <numBlocks, blockSize >> > (d_centerCenterDistDiv2, d_oldcenterCenterDistDiv2, d_s, centers->d_data, centers->d, k, centers->n);
    }
    else {
        innerProd << <numBlocks, blockSize >> > (d_centerCenterDistDiv2, d_s, centers->d_data, centers->d, centers->n);
    }
#else
    // find the inter-center distances
    for (int c1 = 0; c1 < k; ++c1) {
        if (c1 % numThreads == threadId) {
            s[c1] = std::numeric_limits<double>::max();

            for (int c2 = 0; c2 < k; ++c2) {
                // we do not need to consider the case when c1 == c2 as centerCenterDistDiv2[c1*k+c1]
                // is equal to zero from initialization, also this distance should not be used for s[c1]
                if (c1 != c2) {
                    // divide by 2 here since we always use the inter-center
                    // distances divided by 2
                    //std::cout <<sqrt(centerCenterDist2(c1, c2))<< "\n";
                    oldcenterCenterDistDiv2[c1 * k + c2] = centerCenterDistDiv2[c1 * k + c2];
                    centerCenterDistDiv2[c1 * k + c2] = sqrt(centerCenterDist2(c1, c2)) / 2.0;

                    if (centerCenterDistDiv2[c1 * k + c2] < s[c1]) {
                        s[c1] = centerCenterDistDiv2[c1 * k + c2];
                    }
                }
            }
        }
    }
#endif
}

int CO_ElkanKmeans::runThread(int threadId, int maxIterations) {
    iterations = 0;
    
    int startNdx = start(threadId);
    int endNdx = end(threadId);
    unsigned short* closest2 = new unsigned short[endNdx];
    unsigned short* d_closest2;
    auto f = hipMalloc(&d_closest2, endNdx * sizeof(unsigned short));
    if (f != hipSuccess) {
        std::cout << "hipMalloc failed (closest2)" << std::endl;
    }
    
    ub_old = new double[n];
    hipMalloc(&d_ub_old, n * sizeof(double));
    std::fill(ub_old, ub_old + n, std::numeric_limits<double>::max());
    hipMemset(d_lower, 0.0, (n * k) * sizeof(double));

    oldcenterCenterDistDiv2 = new double[k * k];
    hipMalloc(&d_oldcenterCenterDistDiv2, (k*k) * sizeof(double));
    std::fill(oldcenterCenterDistDiv2, oldcenterCenterDistDiv2 + k * k, 0.0);
    hipMemset(d_lower, 0.0, (n * k) * sizeof(double));

    oldcenter2newcenterDis = new double[k * k];
    hipMalloc(&d_oldcenter2newcenterDis, (k * k) * sizeof(double));
    std::fill(oldcenter2newcenterDis, oldcenter2newcenterDis + k * k, 0.0);
    hipMemset(d_lower, 0.0, (n * k) * sizeof(double));

    oldcenters = new double[k * d];
    hipMalloc(&d_oldcenters, (k * d) * sizeof(double));
    std::fill(oldcenters, oldcenters + k * d, 0.0);
    //hipMemset(d_lower, 0.0, (n * k) * sizeof(double));

    hipMalloc(&d_lower, (n * k) * sizeof(double));
    hipMemset(d_lower, 0.0, (n * k) * sizeof(double));

    bool* convergedd = new bool;
    bool* d_converged;
    f = hipMalloc(&d_converged, 1 * sizeof(bool));
    if (f != hipSuccess) {
        std::cout << "hipMalloc failed (converged)" << std::endl;
    }

#if GPUC
    const int nC = endNdx;
    const int blockSizeC = 3 * 32;
    const int numBlocksC = (n + blockSizeC - 1) / blockSizeC;

    const int nM = centers->n;
    const int blockSizeM = 1 * 32;
    const int numBlocksM = (nM + blockSizeM - 1) / blockSizeM;

    hipMemcpy(x->d_data, x->data, (n * d) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_upper, upper, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_ub_old, ub_old, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_assignment, assignment, n * sizeof(unsigned short), hipMemcpyHostToDevice);
    hipMemcpy(centers->d_data, centers->data, (k * d) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_oldcenter2newcenterDis, oldcenter2newcenterDis, (k * k) * sizeof(double), hipMemcpyHostToDevice);
    gpuErrchk(hipMemcpy(sumNewCenters[0]->d_data, sumNewCenters[0]->data, (k * d) * sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_clusterSize, clusterSize[0], k * sizeof(int), hipMemcpyHostToDevice));
#endif

#if GPUC
    while ((iterations < maxIterations) && !(*convergedd)) {
#else
    while ((iterations < maxIterations) && !converged) {
#endif 
        ++iterations;
        *convergedd = true;

        update_center_dists(threadId);
#if GPUC      
        if (iterations > changeIter){
            elkanFunMO << <numBlocksC, blockSizeC >> > (x->d_data, centers->d_data, d_assignment, 
                d_upper, d_s, d_centerCenterDistDiv2, d_oldcenter2newcenterDis, d_oldcenterCenterDistDiv2, d_ub_old, d_centerMovement, k, d, endNdx, d_closest2);
        }
        else{
            elkanFunNoMove << <numBlocksC, blockSizeC >> > (x->d_data, centers->d_data, d_assignment, 
                d_lower, d_upper, d_s, d_centerCenterDistDiv2, k, d, endNdx, d_closest2, 0);
        }

        changeAss << <numBlocksC, blockSizeC >> > (x->d_data, d_assignment, d_closest2, d_clusterSize, sumNewCenters[threadId]->d_data, d, nC, 0);

#else
        for (int i = startNdx; i < endNdx; ++i) {
            unsigned short closest = assignment[i];
            bool r = true;

            if (upper[i] <= s[closest]) {
                continue;
            }

            for (int j = 0; j < k; ++j) {
                if (j == closest) { continue; }
                if (upper[i] <= 2.0 * (oldcenterCenterDistDiv2[assignment[i] * k + j]) - ub_old[i] - centerMovement[j]) { continue; }
                if (upper[i] <= oldcenter2newcenterDis[assignment[i] * k + j] - ub_old[i]) { continue; }  //upper[i] <= lower[i * k + j] ||
                if (upper[i] <= centerCenterDistDiv2[closest * k + j]) { continue; }
#if Countdistance
                numberdistances++;
#endif
                // ELKAN 3(a)
                if (r) {
                    upper[i] = sqrt(pointCenterDist2(i, closest));
                    //lower[i * k + closest] = upper[i];
                    //lower2[i * k + closest] = upper[i];
                    r = false;
                    //if ((upper[i] <= lower[i * k + j]) || (upper[i] <= centerCenterDistDiv2[closest * k + j])) {
                        //continue;
                    //}
                }

                // ELKAN 3(b)
                //lower[i * k + j] = sqrt(pointCenterDist2(i, j));

                if (sqrt(pointCenterDist2(i, j)) < upper[i]) {
                    closest = j;
                    upper[i] = sqrt(pointCenterDist2(i, j));
                }
            }
            if (assignment[i] != closest) {
                changeAssignment(i, closest, threadId);
            }


        }

#if Countdistance
        std::cout << numberdistances << "\n";
#endif

#if Time
        end = std::chrono::system_clock::now();
        elapsed_seconds = end - start;
        std::cout << elapsed_seconds.count() << "\n";
        total_elkan_time += (std::chrono::duration_cast<std::chrono::duration<double>>(std::chrono::high_resolution_clock::now() - start_time));
#endif


        //verifyAssignment(iterations, startNdx, endNdx);

        // ELKAN 4, 5, AND 6
        synchronizeAllThreads();
#endif
#if GPUC 
        /*hipMemcpy(assignment, d_assignment, n * sizeof(unsigned short), hipMemcpyDeviceToHost);
        hipMemcpy(centers->data, centers->d_data, (k * d) * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(x->data, x->d_data, (n * d) * sizeof(double), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        verifyAssignment(iterations, startNdx, endNdx);
        hipMemcpy(d_assignment, assignment, n * sizeof(unsigned short), hipMemcpyHostToDevice);
        hipMemcpy(centers->d_data, centers->data, (k * d) * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(x->d_data, x->data, (n * d) * sizeof(double), hipMemcpyHostToDevice);
        hipDeviceSynchronize();*/

        hipMemcpy(d_converged, convergedd, 1 * sizeof(bool), hipMemcpyHostToDevice);
        elkanMoveCenter << <numBlocksM, blockSizeM >> > (d_centerMovement, d_clusterSize, centers->d_data, sumNewCenters[threadId]->d_data, d_converged, k, d, nM);           
        hipMemcpy(convergedd, d_converged, 1 * sizeof(bool), hipMemcpyDeviceToHost);

        if (iterations > changeIter){        
            const int n = centers->n * centers->n;
            const int blockSize = 1 * 32;
            const int numBlocks = (n + blockSize - 1) / blockSize;
            elkanFBMoveAddition << <numBlocks, blockSize >> > (d_oldcenters, d_oldcenter2newcenterDis, centers->d_data, d, k, centers->n);
        }
#else
        if (threadId == 0) {
            int furthestMovingCenter = move_centers_newbound(oldcenters, oldcenter2newcenterDis);
            converged = (0.0 == centerMovement[furthestMovingCenter]);
        }
#endif          
#if GPUC
        if (!(*convergedd)) {
#else
        if (!converged) {
#endif
            update_bounds(startNdx, endNdx);
        }
        synchronizeAllThreads();

    }
    hipMemcpy(assignment, d_assignment, n * sizeof(unsigned short), hipMemcpyDeviceToHost);
    for (int i = 0; i < 20; i++) {
        std::cout << "assignment: " << assignment[i] << std::endl;
    }
    delete convergedd;
    hipFree(d_converged);
    std::cout << "ITERATIONEN: " << iterations << std::endl;
    return iterations;
}

void CO_ElkanKmeans::update_bounds(int startNdx, int endNdx) {
#if GPUB
    int n = endNdx;
    int blockSize = 3 * 32;
    int numBlocks = (n + blockSize - 1) / blockSize;

    if (iterations > changeIter){  
        updateBoundMO << <numBlocks, blockSize >> > (d_upper, d_ub_old, d_centerMovement, d_assignment, endNdx);
    }
    else{
        updateBound << <numBlocks, blockSize >> > (x->d_data, d_lower, d_upper, d_centerMovement, d_assignment, numLowerBounds, d, k, endNdx);
    }
#else
    for (int i = startNdx; i < endNdx; ++i) {
        ub_old[i] = upper[i];
        upper[i] += centerMovement[assignment[i]];
    }   
    for (int i = startNdx; i < endNdx; ++i) {
        upper[i] += centerMovement[assignment[i]];
    }
#endif
}

void CO_ElkanKmeans::initialize(Dataset const* aX, unsigned short aK, unsigned short* initialAssignment, int aNumThreads) {
    numLowerBounds = aK;
    TriangleInequalityBaseKmeans::initialize(aX, aK, initialAssignment, aNumThreads);
    centerCenterDistDiv2 = new double[k * k];
    hipMalloc(&d_centerCenterDistDiv2, (k * k) * sizeof(double));
    std::fill(centerCenterDistDiv2, centerCenterDistDiv2 + k * k, 0.0);

}

void CO_ElkanKmeans::free() {
    TriangleInequalityBaseKmeans::free();
    hipFree(d_centerCenterDistDiv2);
    hipFree(d_oldcenterCenterDistDiv2);
    hipFree(d_oldcenter2newcenterDis);
    hipFree(d_oldcenters);
    
    delete centerCenterDistDiv2;
    centerCenterDistDiv2 = NULL;
    hipFree(d_ub_old);

    
    delete oldcenterCenterDistDiv2;
    delete oldcenter2newcenterDis;
    delete oldcenters;
    delete ub_old;
    //delete [] oldcenterCenterDistDiv2;
    //oldcenterCenterDistDiv2 = NULL;
    delete centers;
    centers = NULL;
}
int CO_ElkanKmeans::move_centers_newbound(double* oldcenters, double* oldcenter2newcenterDis) {
    int furthestMovingCenter = 0;
    for (int j = 0; j < k; ++j) {
        centerMovement[j] = 0.0;
        int totalClusterSize = 0;
        double old = 0;
        for (int t = 0; t < numThreads; ++t) {
            totalClusterSize += clusterSize[t][j];
        }
        if (totalClusterSize > 0) {
            for (int dim = 0; dim < d; ++dim) {
                double z = 0.0;
                for (int t = 0; t < numThreads; ++t) {
                    z += (*sumNewCenters[t])(j, dim);
                }
                z /= totalClusterSize;
                //std::cout << z << "\n";
                //std::cout << (*centers)(j, dim) << "\n";
                centerMovement[j] += (z - (*centers)(j, dim)) * (z - (*centers)(j, dim));//calculate distance
                //std::cout << (*centers)(j, dim) << "\n";
                old = (*centers)(j, dim);
                //std::cout << (*oldcenters)(j, dim) << "\n";
                oldcenters[j * d + dim] = old;
                //std::cout << (*centers)(j, dim) << "\n";
                (*centers)(j, dim) = z; //update new centers
            }
        }
        centerMovement[j] = sqrt(centerMovement[j]);

        if (centerMovement[furthestMovingCenter] < centerMovement[j]) {
            furthestMovingCenter = j;
        }
    }

    for (int c1 = 0; c1 < k; ++c1) {

        for (int c2 = 0; c2 < k; ++c2)
            if (c1 != c2) {
                oldcenter2newcenterDis[c1 * k + c2] = 0.0;
                for (int dim = 0; dim < d; ++dim) {
                    oldcenter2newcenterDis[c1 * k + c2] += (oldcenters[c1 * d + dim] - (*centers)(c2, dim)) * (oldcenters[c1 * d + dim] - (*centers)(c2, dim));
                }
                oldcenter2newcenterDis[c1 * k + c2] = sqrt(oldcenter2newcenterDis[c1 * k + c2]);
            }
    }

#ifdef COUNT_DISTANCES
    numDistances += k;
#endif

    return furthestMovingCenter;
}