#include "hip/hip_runtime.h"
/* Authors: Greg Hamerly and Jonathan Drake
 * Feedback: hamerly@cs.baylor.edu
 * See: http://cs.baylor.edu/~hamerly/software/kmeans.php
 * Copyright 2014
 */

#include "ham_elkanFB.h"
#include "general_functions.h"
//#include "gpufunctions.h"
#include <cmath>
#include <chrono>

#define GPUA 0
#define GPUB 0
#define GPUC 0

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void HamElkanFB::update_center_dists(int threadId) {
#if GPUA
    const int n = centers->n * centers->n;
    const int blockSize = 1 * 32;
    const int numBlocks = (n + blockSize - 1) / blockSize;

    /* hipMemcpy(centers->d_data, centers->data, (k * d) * sizeof(double), hipMemcpyHostToDevice);*/
    //innerProdFBHam << <numBlocks, blockSize >> > (d_centerCenterDistDiv2, d_s, centers->d_data, centers->d, centers->n);
    hipMemset(d_s, std::numeric_limits<double>::max(), k * sizeof(double));
    innerProd << <numBlocks, blockSize >> > (d_centerCenterDistDiv2, d_s, centers->d_data, centers->d, centers->n);
    /* hipMemcpy(centers->data, centers->d_data, (k * d) * sizeof(double), hipMemcpyDeviceToHost);
     hipMemcpy(centerCenterDistDiv2, d_centerCenterDistDiv2, (k * k) * sizeof(double), hipMemcpyDeviceToHost);
     hipMemcpy(s, d_s, k * sizeof(double), hipMemcpyDeviceToHost);*/

#else

    for (int c1 = 0; c1 < k; ++c1) {
        if (c1 % numThreads == threadId) {
            s[c1] = std::numeric_limits<double>::max();

            for (int c2 = 0; c2 < k; ++c2) {
                if (c1 != c2) {
                    centerCenterDistDiv2[c1 * k + c2] = sqrt(centerCenterDist2(c1, c2)) / 2.0;
                    if (centerCenterDistDiv2[c1 * k + c2] < s[c1]) {
                        s[c1] = centerCenterDistDiv2[c1 * k + c2];
                    }
                }
            }
        }
    }
#endif
}

int HamElkanFB::runThread(int threadId, int maxIterations) {
    int iterations = 0;
    int startNdx = start(threadId);
    int endNdx = end(threadId);

    unsigned short* closest2 = new unsigned short[endNdx];
    unsigned short* d_closest2;
    auto f = hipMalloc(&d_closest2, endNdx * sizeof(unsigned short));
    if (f != hipSuccess) {
        std::cout << "hipMalloc failed (closest2)" << std::endl;
    }

    bool* convergedd = new bool;
    bool* d_converged;
    f = hipMalloc(&d_converged, 1 * sizeof(bool));
    if (f != hipSuccess) {
        std::cout << "hipMalloc failed (converged)" << std::endl;
    }

    double* distances = new double[n * k];
    double* distances2 = new double[n * k];
    hipMalloc(&d_calculated, n * sizeof(bool));
    hipMalloc(&d_distances, (n * k) * sizeof(double));
    //hipMalloc(&d_distances2, (n * k) * sizeof(double));

    converged = false;
    *convergedd = false;

#if GPUC
    hipMemcpy(x->d_data, x->data, (n * d) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_lower, lower, (n * k) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_upper, upper, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_ub_old, ub_old, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_assignment, assignment, n * sizeof(unsigned short), hipMemcpyHostToDevice);
    hipMemcpy(d_oldcenter2newcenterDis, oldcenter2newcenterDis, (k * k) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(centers->d_data, centers->data, (k * d) * sizeof(double), hipMemcpyHostToDevice);
    gpuErrchk(hipMemcpy(sumNewCenters[0]->d_data, sumNewCenters[0]->data, (k * d) * sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_clusterSize, clusterSize[0], k * sizeof(int), hipMemcpyHostToDevice));

    const int nC = endNdx * k;
    std::cout << "nc: " << nC << std::endl;
    const int blockSizeC = 3 * 32;
    const int numBlocksC = (nC + blockSizeC - 1) / blockSizeC;

    const int nD = endNdx;
    const int blockSizeD = 3 * 32;
    const int numBlocksD = (nD + blockSizeD - 1) / blockSizeD;

    const int nM = centers->n;
    const int blockSizeM = 1 * 32;
    const int numBlocksM = (nM + blockSizeM - 1) / blockSizeM;

    unsigned long long int* d_countDistances;
    gpuErrchk(hipMalloc(&d_countDistances, 1 * sizeof(unsigned long long int)));
    hipMemset(d_countDistances, 0, 1 * sizeof(unsigned long long int));

#endif
#if GPUC
    while ((iterations < maxIterations) && !(*convergedd)) {
#else
    while ((iterations < maxIterations) && !converged) {
#endif 
        ++iterations;
        *convergedd = true;

        update_center_dists(threadId);


#if GPUC  
        /*calculateFilter << <numBlocksD, blockSizeD >> > (d_assignment, d_lower, d_upper, d_s, d_maxoldcenter2newcenterDis, d_ub_old, d_calculated, n, d_closest2);
        elkanFunFBHam2TT << <numBlocksC, blockSizeC >> > (x->d_data, centers->d_data, d_distances, d_calculated, k, d, endNdx);
        elkanFunFBHamTT << <numBlocksD, blockSizeD >> > (x->d_data, centers->d_data, d_assignment,
            d_lower, d_upper, d_s, d_centerCenterDistDiv2, d_maxoldcenter2newcenterDis, d_ub_old, k, d, endNdx, d_closest2, d_calculated, d_distances);*/

        //calculateFilter2 << <numBlocksD, blockSizeD >> > (d_assignment, d_lower, d_upper, d_s, d_maxoldcenter2newcenterDis, d_ub_old, d_calculated, n, d_closest2, x->d_data, centers->d_data, d);
       // /*elkanFunFBHam2 << <numBlocksC, blockSizeC >> > (x->d_data, centers->d_data, d_distances, d_calculated, k, d, endNdx*10);
        //elkanFunFBHamBounds << <numBlocksD, blockSizeD >> > (x->d_data, d_lower, d_upper, d_distances, d_calculated, k, d, n, d_closest2);*/

       //// elkanFunFBHam2 << <numBlocksD, blockSizeD >> > (x->d_data, centers->d_data, d_distances2, d_calculated, k, d, endNdx);


       // //hipMemcpy(distances, d_distances, (n * k) * sizeof(double), hipMemcpyDeviceToHost);
       // //hipMemcpy(distances2, d_distances2, (n * k) * sizeof(double), hipMemcpyDeviceToHost);
       // //for (int i = 0; i < n; i++) {
       // //    for (int j = 0; j < k; j++) {
       // //        if (distances[i * k + j] != distances2[i * k + j]) {
       // //            std::cout << "UNGLEICH -- i: " << i << " j: " << j << " " << distances[i * k + j] << " != " << distances2[i * k + j] << std::endl;
       // //        }                
       // //    }
       // //}
       // //hipMemcpy(d_distances, distances, (n * k) * sizeof(double), hipMemcpyHostToDevice);
       // //hipMemcpy(d_distances2, distances2, (n * k) * sizeof(double), hipMemcpyHostToDevice);        

        elkanFunFBHam << <numBlocksD, blockSizeD >> > (x->d_data, centers->d_data, d_assignment,
            d_lower, d_upper, d_s, d_centerCenterDistDiv2, d_maxoldcenter2newcenterDis, d_ub_old, k, d, endNdx, d_closest2, d_countDistances);

            //elkanFunLloyd << <numBlocksD, blockSizeD >> > (x->d_data, centers->d_data, d_assignment, k, d, endNdx, d_closest2);

        changeAss << <numBlocksD, blockSizeD >> > (x->d_data, d_assignment, d_closest2, d_clusterSize, sumNewCenters[threadId]->d_data, d, nC, 0);
#else
        for (int i = startNdx; i < endNdx; ++i) {
            unsigned short closest = assignment[i];
            bool r = true;

            if (upper[i] <= s[closest]) {
                continue;
            }

            for (int j = 0; j < k; ++j) {
                if (j == closest) { continue; }

                if (upper[i] <= lower[i * k + j]) { continue; }
                if (upper[i] <= oldcenter2newcenterDis[assignment[i] * k + j] - ub_old[i]) { continue; }
                if (upper[i] <= centerCenterDistDiv2[closest * k + j]) { continue; }

#if Countdistance
                numberdistances++;
#endif
                // ELKAN 3(a)
                if (r) {
                    upper[i] = sqrt(pointCenterDist2(i, closest));
                    lower[i * k + closest] = upper[i];
                    r = false;
                    if ((upper[i] <= lower[i * k + j]) || (upper[i] <= centerCenterDistDiv2[closest * k + j]) || upper[i] <= oldcenter2newcenterDis[assignment[i] * k + j] - ub_old[i]) {
                        continue;
                    }
                }

                // ELKAN 3(b)
                lower[i * k + j] = sqrt(pointCenterDist2(i, j));

                if (lower[i * k + j] < upper[i]) {
                    closest = j;
                    upper[i] = lower[i * k + j];
                }
            }
            if (assignment[i] != closest) {
                changeAssignment(i, closest, threadId);
            }
        }
#endif


        //verifyAssignment(iterations, startNdx, endNdx);

        // ELKAN 4, 5, AND 6
#if GPUC 
        hipMemcpy(d_converged, convergedd, 1 * sizeof(bool), hipMemcpyHostToDevice);
        elkanMoveCenterFB << <numBlocksM, blockSizeM >> > (d_centerMovement, d_clusterSize, centers->d_data, sumNewCenters[threadId]->d_data, d_oldcenters, d_converged, k, d, nM);
        hipMemcpy(convergedd, d_converged, 1 * sizeof(bool), hipMemcpyDeviceToHost);

        const int n = centers->n * centers->n;
        const int blockSize = 1 * 32;
        const int numBlocks = (n + blockSize - 1) / blockSize;
        hipMemset(d_oldcenter2newcenterDis, 0.0, (k * k) * sizeof(double));
        elkanFBMoveAddition << <numBlocks, blockSize >> > (d_oldcenters, d_oldcenter2newcenterDis, centers->d_data, d, k, centers->n);
        elkanFBMoveAdditionHam << <centers->n, 1 >> > (d_oldcenters, d_oldcenter2newcenterDis, d_maxoldcenter2newcenterDis, k, centers->n);
#else
        int furthestMovingCenter = move_centers_newbound(oldcenters, oldcenter2newcenterDis);
        converged = (0.0 == centerMovement[furthestMovingCenter]);

#endif
#if GPUC
        if (!(*convergedd)) {
#else
        if (!converged) {
#endif
            update_bounds(startNdx, endNdx);
        }
        }
    /* hipMemcpy(assignment, d_assignment, n * sizeof(unsigned short), hipMemcpyDeviceToHost);
     for (int i = 0; i < 20; i++) {
         std::cout << "assignment: " << assignment[i] << std::endl;
     }*/

    std::cout << "ITERATIONEN: " << iterations << std::endl;
    return iterations;
    }

void HamElkanFB::update_bounds(int startNdx, int endNdx) {
#if GPUB
    int n = endNdx;
    int blockSize = 3 * 32;
    int numBlocks = (n + blockSize - 1) / blockSize;

    /*hipMemcpy(d_lower, lower, (n * k) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_upper, upper, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_ub_old, ub_old, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_assignment, assignment, n * sizeof(unsigned short), hipMemcpyHostToDevice);
    hipMemcpy(d_centerMovement, centerMovement, k * sizeof(double), hipMemcpyHostToDevice);*/

    updateBoundFBHam << <numBlocks, blockSize >> > (d_lower, d_upper, d_ub_old, d_centerMovement, d_assignment, numLowerBounds, k, endNdx);

    /*hipMemcpy(lower, d_lower, (n * k) * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(upper, d_upper, n * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(ub_old, d_ub_old, n * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(assignment, d_assignment, n * sizeof(unsigned short), hipMemcpyDeviceToHost);
    hipMemcpy(centerMovement, d_centerMovement, k * sizeof(double), hipMemcpyDeviceToHost);*/

#else
    for (int i = startNdx; i < endNdx; ++i) {
        ub_old[i] = upper[i];
    }

    for (int i = startNdx; i < endNdx; ++i) {
        upper[i] += centerMovement[assignment[i]];
        for (int j = 0; j < k; ++j) {
            lower[i * numLowerBounds + j] -= centerMovement[j];
        }
    }
#endif


}

void HamElkanFB::initialize(Dataset const* aX, unsigned short aK, unsigned short* initialAssignment, int aNumThreads) {
    numLowerBounds = aK;
    TriangleInequalityBaseKmeans::initialize(aX, aK, initialAssignment, aNumThreads);

    centerCenterDistDiv2 = new double[k * k];
    hipMalloc(&d_centerCenterDistDiv2, (k * k) * sizeof(double));
    std::fill(centerCenterDistDiv2, centerCenterDistDiv2 + k * k, 0.0);
    oldcenter2newcenterDis = new double[k * k];
    hipMalloc(&d_oldcenter2newcenterDis, (k * k) * sizeof(double));
    std::fill(oldcenter2newcenterDis, oldcenter2newcenterDis + k * k, 0.0);
    hipMalloc(&d_maxoldcenter2newcenterDis, k * sizeof(double));
    ub_old = new double[n];
    hipMalloc(&d_ub_old, n * sizeof(double));
    std::fill(ub_old, ub_old + n, std::numeric_limits<double>::max());
    lower = new double[n];
    hipMalloc(&d_lower, n * sizeof(double));
    std::fill(lower, lower + n, 0.0);
    oldcenters = new double[k * d];
    hipMalloc(&d_oldcenters, (k * d) * sizeof(double));
    std::fill(oldcenters, oldcenters + k * d, 0.0);
}

void HamElkanFB::free() {
    TriangleInequalityBaseKmeans::free();
    hipFree(d_centerCenterDistDiv2);
    hipFree(d_lower);
    hipFree(d_ub_old);
    hipFree(d_oldcenters);
    hipFree(d_oldcenter2newcenterDis);
    hipFree(d_maxoldcenter2newcenterDis);
    delete centerCenterDistDiv2;
    delete lower;
    delete ub_old;
    delete oldcenters;
    //delete[] centerCenterDistDiv2;
    //centerCenterDistDiv2 = NULL;
    //delete [] oldcenterCenterDistDiv2;
    //oldcenterCenterDistDiv2 = NULL;
    delete centers;
    centers = NULL;
}

int HamElkanFB::move_centers_newbound(double* oldcenters, double* oldcenter2newcenterDis) {

    int furthestMovingCenter = 0;
    for (int j = 0; j < k; ++j) {
        centerMovement[j] = 0.0;
        int totalClusterSize = 0;
        for (int t = 0; t < numThreads; ++t) {
            totalClusterSize += clusterSize[t][j];
        }
        if (totalClusterSize > 0) {
            for (int dim = 0; dim < d; ++dim) {
                double z = 0.0;
                for (int t = 0; t < numThreads; ++t) {
                    z += (*sumNewCenters[t])(j, dim);
                }
                z /= totalClusterSize;
                centerMovement[j] += (z - (*centers)(j, dim)) * (z - (*centers)(j, dim));//calculate distance
                oldcenters[j * d + dim] = (*centers)(j, dim);
                (*centers)(j, dim) = z;
            }
        }
        centerMovement[j] = sqrt(centerMovement[j]);

        if (centerMovement[furthestMovingCenter] < centerMovement[j]) {
            furthestMovingCenter = j;
        }
    }

    for (int c1 = 0; c1 < k; ++c1) {
        for (int c2 = 0; c2 < k; ++c2)
            if (c1 != c2) {
                oldcenter2newcenterDis[c1 * k + c2] = 0.0;
                for (int dim = 0; dim < d; ++dim) {
                    oldcenter2newcenterDis[c1 * k + c2] += (oldcenters[c1 * d + dim] - (*centers)(c2, dim)) * (oldcenters[c1 * d + dim] - (*centers)(c2, dim));
                }
                oldcenter2newcenterDis[c1 * k + c2] = sqrt(oldcenter2newcenterDis[c1 * k + c2]);
            }
    }
    //return 0;
    return furthestMovingCenter;
}
